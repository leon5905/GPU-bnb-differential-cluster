#include "hip/hip_runtime.h"
#include "kernel_trifle.cuh"
#include <iostream>
#include <cstring>

namespace TRIFLE{
    /*
	* BC specific permutation and DTT
	*/
    //Contains configuration (macro / c++ global variable) intended to be used across different translation unit
    __shared__ unsigned long long perm_lookup_shared[32][16][2]; 
    __device__ unsigned long long perm_lookup_global[32][16][2];
    __device__ unsigned long long perm_lookup_global_reversed[32][16][2];
    
    unsigned char perm_host[128];
    unsigned char perm_host_reversed[128];

    unsigned long long perm_lookup_host[32][16][2]; //8192 bytes, 8KB, one SM can have 49KB should be fine
    unsigned long long perm_lookup_host_reversed[32][16][2];
    //_host  //ONLY USED in main class || not used here

    //NOTE: _host have no uses inside this class
    __shared__ unsigned int diff_table_shared[16][8];  //NOTE: init in kernel by 1st thread of the block.
    __device__ unsigned int diff_table_global[][8] = {
        {0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0},
        {0xc, 0x6, 0x7, 0x8, 0xa, 0xd, 0xe, 0x0},
        {0x9, 0x1, 0x5, 0xb, 0xc, 0xd, 0xe, 0x0},
        {0x7, 0x1, 0x2, 0x4, 0x5, 0x9, 0xb, 0x0},
        {0x3, 0x2, 0x7, 0x9, 0xa, 0xb, 0xd, 0x0},
        {0x5, 0x6, 0x7, 0x9, 0xa, 0xd, 0xf, 0x0},
        {0xe, 0x2, 0x3, 0x4, 0x7, 0x8, 0xa, 0x0},
        {0x4, 0x2, 0x3, 0x9, 0xb, 0xc, 0xf, 0x0},
        {0x6, 0x3, 0x4, 0x5, 0x7, 0xb, 0xe, 0x0},
        {0xb, 0x1, 0x2, 0x8, 0xa, 0xc, 0xd, 0x0},
        {0xa, 0x3, 0x5, 0xb, 0xc, 0xe, 0xf, 0x0},
        {0x2, 0x1, 0x6, 0x9, 0xc, 0xd, 0xf, 0x0},
        {0xd, 0x1, 0x4, 0x5, 0x6, 0x8, 0xe, 0x0},
        {0x1, 0x3, 0x6, 0x8, 0xc, 0xe, 0xf, 0x0},
        {0x8, 0x3, 0x4, 0x6, 0x7, 0x9, 0xf, 0x0},
        {0xf, 0x1, 0x2, 0x4, 0x5, 0x8, 0xa, 0x0},
    };
    __device__ unsigned int diff_table_global_reversed[][8] = {
        { 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0},
        { 0xd, 0x2, 0x3, 0x9, 0xb, 0xc, 0xf, 0x0},
        { 0xb, 0x3, 0x4, 0x6, 0x7, 0x9, 0xf, 0x0},
        { 0x4, 0x6, 0x7, 0x8, 0xa, 0xd, 0xe, 0x0},
        { 0x7, 0x3, 0x6, 0x8, 0xc, 0xe, 0xf, 0x0},
        { 0x5, 0x2, 0x3, 0x8, 0xa, 0xc, 0xf, 0x0},
        { 0x8, 0x1, 0x5, 0xb, 0xc, 0xd, 0xe, 0x0},
        { 0x3, 0x1, 0x4, 0x5, 0x6, 0x8, 0xe, 0x0},
        { 0xe, 0x1, 0x6, 0x9, 0xc, 0xd, 0xf, 0x0},
        { 0x2, 0x3, 0x4, 0x5, 0x7, 0xb, 0xe, 0x0},
        { 0xa, 0x1, 0x4, 0x5, 0x6, 0x9, 0xf, 0x0},
        { 0x9, 0x2, 0x3, 0x4, 0x7, 0x8, 0xa, 0x0},
        { 0x1, 0x2, 0x7, 0x9, 0xa, 0xb, 0xd, 0x0},
        { 0xc, 0x1, 0x2, 0x4, 0x5, 0x9, 0xb, 0x0},
        { 0x6, 0x1, 0x2, 0x8, 0xa, 0xc, 0xd, 0x0},
        { 0xf, 0x5, 0x7, 0xa, 0xb, 0xd, 0xe, 0x0}
    };
    unsigned int diff_table_host[][8] = {
        {0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0},
        {0xc, 0x6, 0x7, 0x8, 0xa, 0xd, 0xe, 0x0},
        {0x9, 0x1, 0x5, 0xb, 0xc, 0xd, 0xe, 0x0},
        {0x7, 0x1, 0x2, 0x4, 0x5, 0x9, 0xb, 0x0},
        {0x3, 0x2, 0x7, 0x9, 0xa, 0xb, 0xd, 0x0},
        {0x5, 0x6, 0x7, 0x9, 0xa, 0xd, 0xf, 0x0},
        {0xe, 0x2, 0x3, 0x4, 0x7, 0x8, 0xa, 0x0},
        {0x4, 0x2, 0x3, 0x9, 0xb, 0xc, 0xf, 0x0},
        {0x6, 0x3, 0x4, 0x5, 0x7, 0xb, 0xe, 0x0},
        {0xb, 0x1, 0x2, 0x8, 0xa, 0xc, 0xd, 0x0},
        {0xa, 0x3, 0x5, 0xb, 0xc, 0xe, 0xf, 0x0},
        {0x2, 0x1, 0x6, 0x9, 0xc, 0xd, 0xf, 0x0},
        {0xd, 0x1, 0x4, 0x5, 0x6, 0x8, 0xe, 0x0},
        {0x1, 0x3, 0x6, 0x8, 0xc, 0xe, 0xf, 0x0},
        {0x8, 0x3, 0x4, 0x6, 0x7, 0x9, 0xf, 0x0},
        {0xf, 0x1, 0x2, 0x4, 0x5, 0x8, 0xa, 0x0},
    };
    //Init by init
    unsigned int diff_table_host_reversed[][8] = {
        { 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0},
        { 0xd, 0x2, 0x3, 0x9, 0xb, 0xc, 0xf, 0x0},
        { 0xb, 0x3, 0x4, 0x6, 0x7, 0x9, 0xf, 0x0},
        { 0x4, 0x6, 0x7, 0x8, 0xa, 0xd, 0xe, 0x0},
        { 0x7, 0x3, 0x6, 0x8, 0xc, 0xe, 0xf, 0x0},
        { 0x5, 0x2, 0x3, 0x8, 0xa, 0xc, 0xf, 0x0},
        { 0x8, 0x1, 0x5, 0xb, 0xc, 0xd, 0xe, 0x0},
        { 0x3, 0x1, 0x4, 0x5, 0x6, 0x8, 0xe, 0x0},
        { 0xe, 0x1, 0x6, 0x9, 0xc, 0xd, 0xf, 0x0},
        { 0x2, 0x3, 0x4, 0x5, 0x7, 0xb, 0xe, 0x0},
        { 0xa, 0x1, 0x4, 0x5, 0x6, 0x9, 0xf, 0x0},
        { 0x9, 0x2, 0x3, 0x4, 0x7, 0x8, 0xa, 0x0},
        { 0x1, 0x2, 0x7, 0x9, 0xa, 0xb, 0xd, 0x0},
        { 0xc, 0x1, 0x2, 0x4, 0x5, 0x9, 0xb, 0x0},
        { 0x6, 0x1, 0x2, 0x8, 0xa, 0xc, 0xd, 0x0},
        { 0xf, 0x5, 0x7, 0xa, 0xb, 0xd, 0xe, 0x0}
    };

    __shared__ float prob_table_shared[16][8];  //NOTE: init in kernel by 1st thread of the block.
    float prob_table_host[16][8]={
        {1, 1, 1, 1, 1, 1, 1, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
        {0.25f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 0.125f, 1},
    };
    __device__ unsigned int freq_table_global[][8] = {
        {16, 16, 16, 16, 16, 16, 16, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
    };
    unsigned int freq_table_host[][8] = {
        {16, 16, 16, 16, 16, 16, 16, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
        {4, 2, 2, 2, 2, 2, 2, 16},
    };

    __shared__ unsigned int diff_table_size_shared[16];  //NOTE: init in kernel by 1st thread of the block.
    __device__ unsigned int diff_table_size_global[16] = {1, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7};
    unsigned int diff_table_size_host[16] = {1, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7};  //ONLY USED in main class || not used here

    /*
    * DX and DY changes
    */
    //NOTE: change the following
    //Refernce Value for PATTERN_ROUND = 10
    //Dx -> Dy : 7000 0000 0000 0000 0000 0000 0000 0000   ->   4000 0000 4000 0000 0000 0000 0000 0000 (before permutation is line below)
    //DY_below is 0c00 0000 0000 0000 0000 0000 0000 0000
    //Cluster Probabilities:-27.9928
    //Number of Cluster Trails : 31
    //HACK: CHANGE the folloig value together with final_dy
    //NOTE: IT IS DY_B4Permutation... (save performance on GPU)

	//Constant memory because it is accessed by the same warp @ the same addresses. (broadcasting) else request will be serialized
    __constant__ unsigned char final_dy_constant[32] = {
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x6, 0x0, 0x6, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x6, 0x0, 0x6, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0
	};
    unsigned char final_dy_host[32] = {
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x6, 0x0, 0x6, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x6, 0x0, 0x6, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0
	};

    unsigned char ref_dx_host[32] = {
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0xb,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0,
        0x0, 0x0, 0x0, 0x0
    };

    __constant__ float CLUSTER_PROB_BOUND_const = 0; 
};

void TRIFLE::init(){
    std::cout <<"\nInit Trifle Reverse Differential Table:{\n";

    std::cout <<"\nTrifle Permutation:{\n";
    for (int i = 0; i < 128; i++) {
        TRIFLE::perm_host[i] = (i / 4) + ((i % 4) * 32);
        std::cout << (int) perm_host[i]<< ",";
    }
    std::cout << "\n}\n" ;

    std::cout <<"\nTrifle Permutation Reversed:{\n";
    for (int i=0;i<128;i++){
        TRIFLE::perm_host_reversed[perm_host[i]] = i;
    }
    for (int i=0;i<128;i++){
        std::cout << (int) perm_host_reversed[i]<< ",";
    }
    std::cout << "}\n" ;

    //--
    std::cout <<"\n4bit Permutation LUTable * 32 (Size is 32*16*16 is 8192Bytes) :{\n";
    for (int sbox_pos=0;sbox_pos<32;sbox_pos++){
        for (int sbox_val=0;sbox_val<16;sbox_val++){
            unsigned char dx[32] = {0};
            dx[sbox_pos] = sbox_val;

            //Do permutation
            unsigned long long front_64 = 0, back_64 = 0, front_64_reversed=0, back_64_reversed=0;
			for (int i = 0; i < 32; i++) {
				if (dx[i] > 0) {
					for (int j = 0; j < 4; j++) {
                        //Actually filtered_bit
						unsigned long long filtered_word = ((dx[i] & (0x1 << j)) >> j) & 0x1;
						if (filtered_word == 0) continue; //no point continue if zero, go to next elements

                        int bit_pos = (TRIFLE::perm_host[((31 - i) * 4) + j]);
                        int bit_pos_reversed = (TRIFLE::perm_host_reversed[((31 - i) * 4) + j]);

						if ((bit_pos / 64) == 1) {  //Front
							bit_pos -= 64;
							front_64 |= (filtered_word << bit_pos);
						}
						else {  //Back
							back_64 |= (filtered_word << bit_pos);
                        }
                        
                        if ((bit_pos_reversed / 64) == 1) {  //Front
							bit_pos_reversed -= 64;
							front_64_reversed |= (filtered_word << bit_pos_reversed);
						}
						else {  //Back
							back_64_reversed |= (filtered_word << bit_pos_reversed);
                        }
					}
				}
			}
            
            //Front 64, 0-15, Back64 - 16-31
            perm_lookup_host[sbox_pos][sbox_val][0]=front_64;
            perm_lookup_host[sbox_pos][sbox_val][1]=back_64;

            perm_lookup_host_reversed[sbox_pos][sbox_val][0]=front_64_reversed;
            perm_lookup_host_reversed[sbox_pos][sbox_val][1]=back_64_reversed;
        }
    }
    std::cout << "}\n" ;
    
    std::cout << "\nTransfered constant matsui bound from host to device";
    auto cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(TRIFLE::CLUSTER_PROB_BOUND_const), &CLUSTER_PROB_BOUND, sizeof(float));
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol CLUSTER_PROB_BOUND_const failed!");
		goto Error;
	}

    std::cout << "\nTransfered perm_LUhost from host to device";
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(TRIFLE::perm_lookup_global), TRIFLE::perm_lookup_host, sizeof(unsigned long long)*32*16*2);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy perm_LUhost failed!");
		goto Error;
    }
    
    std::cout << "\nTransfered perm_LUhost Reversed from host to device";
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(TRIFLE::perm_lookup_global_reversed), TRIFLE::perm_lookup_host_reversed, sizeof(unsigned long long)*32*16*2);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy perm_LUhost failed!");
		goto Error;
	}

    std::cout <<"\n----\n";
    return;

Error:
    std::cout << "\nCritical Error. Aborting Program";
    if (cudaStatus != hipSuccess) {
        hipError_t err = hipGetLastError();
        std::cout << "\nCRITICAL ERROR from TRIFLE init...";
        fprintf(stderr, "\nError Code %d : %s: %s .", cudaStatus, hipGetErrorName(err), hipGetErrorString(err));
        std::cout << "\nPress any key to continue...";
        getchar();
    }
};

//TODO refactor cuda error check to inline function
Kernel_TRIFLE_t::Kernel_TRIFLE_t(int thread_id, unsigned char *& pinned_host_dx_rounds, float *&cur_round_prob_pinned, int *&next_round_sbox_num_and_index,
    unsigned char *&pinned_input_dx, int *&pinned_input_sbox_index){
    
    hipStreamCreate( &(this->stream_obj) );

    int round_to_allocate = PATTERN_ROUND_MITM_FORWARD > PATTERN_ROUND_MITM_BACKWARD? PATTERN_ROUND_MITM_FORWARD : PATTERN_ROUND_MITM_BACKWARD;
    if (round_to_allocate < 20)
        round_to_allocate = 20;

    //CUDA specific init
    //Should use thread id to select devices...
    auto cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!");
		goto Error;
	}

    //CUDA ALLOC (Internal Use)
    //Round Output
    cudaStatus = hipMalloc((void**)& device_prob_arr, sizeof(float)*MAX_BRANCH_PER_ROUND);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  device_prob_arr @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(device_prob_arr, 0, sizeof(float) * MAX_BRANCH_PER_ROUND);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset device_prob_arr failed!");
		goto Error;
	}
    
    cudaStatus = hipMalloc((void**)& device_dy_arr, sizeof(unsigned char)* 32 * MAX_BRANCH_PER_ROUND);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  device_dy_arr @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(device_dy_arr, 0, sizeof(unsigned char)* 32 * MAX_BRANCH_PER_ROUND);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset device_dy_arr failed!");
		goto Error;
    }

    cudaStatus = hipMalloc((void**)& device_sbox_num_and_index_arr, sizeof(int)* 9 * MAX_BRANCH_PER_ROUND);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  device_sbox_num_and_index_arr @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(device_sbox_num_and_index_arr, 0, sizeof(int)* 9 * MAX_BRANCH_PER_ROUND);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset device_sbox_num_and_index_arr failed!");
		goto Error;
    }

    //Round Input
    cudaStatus = hipMalloc((void**)& device_dx, sizeof(unsigned char)* 32);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  device_dx @init failed!");
		goto Error;
    }

    cudaStatus = hipMalloc((void**)& device_sbox_index, sizeof(int)* 8);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  device_sbox_index @init failed!");
		goto Error;
    }

    //Reduction Output
    cudaStatus = hipMalloc((void**)& device_cluster_size_final, sizeof(unsigned int)* THREAD_PER_BLOCK * BLOCK_NUM);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  device_cluster_size_final @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(device_cluster_size_final, 0, sizeof(unsigned int)* THREAD_PER_BLOCK * BLOCK_NUM);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset device_cluster_size_final failed!");
		goto Error;
	}

    cudaStatus = hipMalloc((void**)& device_prob_final, sizeof(float)*  THREAD_PER_BLOCK * BLOCK_NUM);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  device_prob_final @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(device_prob_final, 0, sizeof(float)*  THREAD_PER_BLOCK * BLOCK_NUM);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset device_prob_final failed!");
		goto Error;
    }

    //MITM 
    cudaStatus = hipMalloc((void**)& MITM_prob_interm_global, sizeof(float)*  Kernel_TRIFLE_t::MITM_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  MITM_prob_interm_global @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(MITM_prob_interm_global, 0, sizeof(float)*  Kernel_TRIFLE_t::MITM_size );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_prob_interm_global failed!");
		goto Error;
    }

    cudaStatus = hipMalloc((void**)& MITM_size_interm_global, sizeof(int)*  Kernel_TRIFLE_t::MITM_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  MITM_prob_interm_global @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(MITM_size_interm_global, 0, sizeof(int)*  Kernel_TRIFLE_t::MITM_size );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_prob_interm_global failed!");
		goto Error;
    }

    cudaStatus = hipMalloc((void**)& MITM_prob_final_global, sizeof(double)*  THREAD_PER_BLOCK * BLOCK_NUM );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  MITM_prob_interm_global @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(MITM_prob_final_global, 0, sizeof(double)*  THREAD_PER_BLOCK * BLOCK_NUM );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_prob_interm_global failed!");
		goto Error;
    }

    cudaStatus = hipMalloc((void**)& MITM_size_final_global, sizeof(long long)*  THREAD_PER_BLOCK * BLOCK_NUM );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc  MITM_size_final_global @init failed!");
		goto Error;
    }
    cudaStatus = hipMemset(MITM_size_final_global, 0, sizeof(long long)*  THREAD_PER_BLOCK * BLOCK_NUM );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_size_final_global failed!");
		goto Error;
    }

    //CUDA HOST ALLOC (External Coordination)
    //22 rounds maximum (LAST ROUND IS NOT STORED but reduction)

    //Pinned Device->Host output
    cudaStatus = hipHostAlloc((void**)& pinned_host_dx_rounds, sizeof(unsigned char)*32*MAX_BRANCH_PER_ROUND*(round_to_allocate-1), hipHostMallocDefault);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudahostMalloc failed!");
		goto Error;
    }
    
    cudaStatus = hipHostAlloc((void**)& cur_round_prob_pinned, sizeof(float)*MAX_BRANCH_PER_ROUND*(round_to_allocate-1), hipHostMallocDefault);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudahostMalloc failed!");
		goto Error;
    }

    cudaStatus = hipHostAlloc((void**)& next_round_sbox_num_and_index, sizeof(int)*9*MAX_BRANCH_PER_ROUND*(round_to_allocate-1), hipHostMallocDefault);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudahostMalloc failed!");
		goto Error;
    }
    memset(next_round_sbox_num_and_index,0, sizeof(int)*9*MAX_BRANCH_PER_ROUND*(round_to_allocate-1));

    //Pinned Host->Device Input
    cudaStatus = hipHostAlloc((void**)& pinned_input_dx, sizeof(unsigned char)*32, hipHostMallocDefault);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudahostMalloc failed!");
		goto Error;
    }
    cudaStatus = hipHostAlloc((void**)& pinned_input_sbox_index, sizeof(int)*8, hipHostMallocDefault);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudahostMalloc failed!");
		goto Error;
    }

    return;

    Error:
    std::cout << "\nCritical Error. Aborting Program";
	if (cudaStatus != hipSuccess) {
		hipError_t err = hipGetLastError();
		std::cout << "\nCRITICAL ERROR...";
		fprintf(stderr, "\nError Code %d : %s: %s .", cudaStatus, hipGetErrorName(err), hipGetErrorString(err));
		std::cout << "\nPress any key to continue...";
		getchar();
	}
};

//One round kernel compute for 0 -> n-1, Last 2 parameter is both hostalloc and cudalloc
__global__ void kernel_trifle_n(unsigned char* dx, int* sbox_num, int* sbox_index, float* cur_prob, float* dev_new_prob_arr,  unsigned int* dev_cluster_size_arr){
     //Number of threads must be >=32
    if (threadIdx.x <32){
        if (threadIdx.x<16){
            TRIFLE::diff_table_size_shared[threadIdx.x] = TRIFLE::diff_table_size_global[threadIdx.x];

            for (int j = 0; j < 8; j++) {
                TRIFLE::diff_table_shared[threadIdx.x][j] = TRIFLE::diff_table_global[threadIdx.x][j];
                TRIFLE::prob_table_shared[threadIdx.x][j] = TRIFLE::freq_table_global[threadIdx.x][j]/16.0; 
            }
        }

        for (int j=0;j<16;j++){
            TRIFLE::perm_lookup_shared[threadIdx.x][j][0]= TRIFLE::perm_lookup_global[threadIdx.x][j][0];
            TRIFLE::perm_lookup_shared[threadIdx.x][j][1]= TRIFLE::perm_lookup_global[threadIdx.x][j][1];
        }
    }

    __syncthreads(); //wait for init to be finished, sync up all threads within a block... shared memory lies within each block.

	//Computing target array index (id and output_range)
    //I - THREAD ID / total thread (including all block) - Used to coordinate splitting of tasks
    int thread_id_global = (blockIdx.x * THREAD_PER_BLOCK) + threadIdx.x; 
	float prob_thread = (*cur_prob); //Caching into local memory
    int sbox_num_local = (*sbox_num); //NOTE: difference from n-1, where parameter is not pointer
    
    //Start Range
	// 2560 different threads  1-2559
	//Each thread is responsible for 2401 array entry in result array (n-1 rounds 2 rounds results)
    //There will be 256*10 = 2560 threads handling 2401 

    //0 - 2400
    int thread_id_workforce = 2401;
	if (thread_id_global >= thread_id_workforce) {
		return;
    } 

	//1 Round for 4AS and above
	unsigned char cur_thread_partial_dy[32] = { 0 };
	//Subs for 4 AS
    int divide_factor = 1;
    int diff_freq_index;
    int remaining_value = thread_id_global; //0 - 2400

    //NOTE: Only support sbox>=4
    for (int i = 0; i < 4; i++) {
        diff_freq_index = (remaining_value / divide_factor) % 7; 

        unsigned char cur_val = dx[sbox_index[i]];

        cur_thread_partial_dy[sbox_index[i]] = TRIFLE::diff_table_shared[cur_val][diff_freq_index]; //Assigning target val to partial_dy

        prob_thread *= (TRIFLE::prob_table_shared[cur_val][diff_freq_index]);

        divide_factor *= 7;
    }
    
    if (sbox_num_local >  4){
        int s6,s7,s8;
        if (sbox_num_local == 5){
            s6 = 0;
            s7 = 0; 
            s8 = 0;
        }
        else if (sbox_num_local == 6){
            s6 = 6;
            s7 = 0; 
            s8 = 0;
        }
        else if (sbox_num_local==7){
            s6 = 6;
            s7 = 6; 
            s8 = 0;
        }
        else{
            s6 = 6;
            s7 = 6; 
            s8 = 6;
        }

        for (int s8_loop_i=0; s8_loop_i <= s8; s8_loop_i++){
            for (int i=4;i<sbox_num_local;i++){
                cur_thread_partial_dy[sbox_index[i]] = dx[sbox_index[i]];
            }    

            float cur_prob_final_fourth = prob_thread;
            unsigned char cur_thread_partial_dy_RN_fourth[33]; //The [32] or size 33 is for fast procesing only
            memcpy(cur_thread_partial_dy_RN_fourth, cur_thread_partial_dy, 32);
            cur_thread_partial_dy_RN_fourth[32] = 0;
    
            unsigned char cur_val = cur_thread_partial_dy_RN_fourth[sbox_index[7]];
            cur_thread_partial_dy_RN_fourth[sbox_index[7]] = TRIFLE::diff_table_shared[cur_val][s8_loop_i];
            cur_prob_final_fourth *= (TRIFLE::prob_table_shared[cur_val][s8_loop_i]);
        
            for (int s7_loop_i=0; s7_loop_i <= s7; s7_loop_i++){
                float cur_prob_final_third = cur_prob_final_fourth;
                unsigned char cur_thread_partial_dy_RN_third[33]; //The [32] or size 33 is for fast procesing only
                memcpy(cur_thread_partial_dy_RN_third, cur_thread_partial_dy_RN_fourth, 33);
                //cur_thread_partial_dy_RN_third[32] = 0; // No need as already copied over
    
                cur_val = cur_thread_partial_dy_RN_third[sbox_index[6]];
                cur_thread_partial_dy_RN_third[sbox_index[6]] = TRIFLE::diff_table_shared[cur_val][s7_loop_i];
                cur_prob_final_third *= (TRIFLE::prob_table_shared[cur_val][s7_loop_i]);

                for (int s6_loop_i=0; s6_loop_i <= s6; s6_loop_i++){
                    float cur_prob_final_second = cur_prob_final_third;
                    unsigned char cur_thread_partial_dy_RN_second[33]; //The [32] or size 33 is for fast procesing only
                    memcpy(cur_thread_partial_dy_RN_second, cur_thread_partial_dy_RN_third, 33);
    
                    cur_val = cur_thread_partial_dy_RN_second[sbox_index[5]];
                    cur_thread_partial_dy_RN_second[sbox_index[5]] = TRIFLE::diff_table_shared[cur_val][s6_loop_i];
                    cur_prob_final_second *= (TRIFLE::prob_table_shared[cur_val][s6_loop_i]);

                    for (int s5_loop_i=0;s5_loop_i<7;s5_loop_i++){
                        float cur_prob_final = cur_prob_final_second;
                        unsigned char cur_thread_partial_dy_RN[33]; //The [32] or size 33 is for fast procesing only
                        memcpy(cur_thread_partial_dy_RN, cur_thread_partial_dy_RN_second, 33);
    
                        //Substitution final
                        cur_val = cur_thread_partial_dy_RN[sbox_index[4]];
                        cur_thread_partial_dy_RN[sbox_index[4]] = TRIFLE::diff_table_shared[cur_val][s5_loop_i];
                        cur_prob_final *= (TRIFLE::prob_table_shared[cur_val][s5_loop_i] );    

                        bool is_same = true;
                        for (int i=0;i<32;i++){
                            if (TRIFLE::final_dy_constant[i] != cur_thread_partial_dy_RN[i]){
                                is_same= false;
                                break;
                            }
                        }
                    
                        if (is_same){
                            dev_new_prob_arr[thread_id_global] += cur_prob_final;
                            dev_cluster_size_arr[thread_id_global] += 1;
                        }
                    }
                }
            }
        }
    }
    else{
        //Calculate whether to add to final dx dy
        bool is_same = true;
        for (int i=0;i<32;i++){
            if (TRIFLE::final_dy_constant[i] != cur_thread_partial_dy[i]){
                is_same= false;
                break;
            }
        }

        if (is_same){
            dev_new_prob_arr[thread_id_global] += prob_thread;
            dev_cluster_size_arr[thread_id_global] += 1;
        }
    }
};

__global__ void kernel_trifle_n_forward(unsigned char* dx, int* sbox_num, int* sbox_index, float* cur_prob, float* dev_new_prob_arr,  unsigned int* dev_cluster_size_arr,
    float* MITM_prob_interm_global, int* MITM_size_interm_global){
    //Number of threads must be >=32
   if (threadIdx.x <32){
       if (threadIdx.x<16){
           TRIFLE::diff_table_size_shared[threadIdx.x] = TRIFLE::diff_table_size_global[threadIdx.x];

           for (int j = 0; j < 8; j++) {
               TRIFLE::diff_table_shared[threadIdx.x][j] = TRIFLE::diff_table_global[threadIdx.x][j];
               TRIFLE::prob_table_shared[threadIdx.x][j] = TRIFLE::freq_table_global[threadIdx.x][j]/16.0; 
           }
       }

       for (int j=0;j<16;j++){
           TRIFLE::perm_lookup_shared[threadIdx.x][j][0]= TRIFLE::perm_lookup_global[threadIdx.x][j][0];
           TRIFLE::perm_lookup_shared[threadIdx.x][j][1]= TRIFLE::perm_lookup_global[threadIdx.x][j][1];
       }
   }

   __syncthreads(); //wait for init to be finished, sync up all threads within a block... shared memory lies within each block.

   //Computing target array index (id and output_range)
   //I - THREAD ID / total thread (including all block) - Used to coordinate splitting of tasks
   int thread_id_global = (blockIdx.x * THREAD_PER_BLOCK) + threadIdx.x; 
   float prob_thread = (*cur_prob); //Caching into local memory
   int sbox_num_local = (*sbox_num); //NOTE: difference from n-1, where parameter is not pointer
   
   //Start Range
   // 2560 different threads  1-2559
   //Each thread is responsible for 2401 array entry in result array (n-1 rounds 2 rounds results)
   //There will be 256*10 = 2560 threads handling 2401 

   //0 - 2400
   int thread_id_workforce = 2401;
   if (thread_id_global >= thread_id_workforce) {
       return;
   } 

   //1 Round for 4AS and above
   unsigned char cur_thread_partial_dy[32] = { 0 };
   //Subs for 4 AS
   int divide_factor = 1;
   int diff_freq_index;
   int remaining_value = thread_id_global; //0 - 2400

   //NOTE: Only support sbox>=4
   for (int i = 0; i < 4; i++) {
       diff_freq_index = (remaining_value / divide_factor) % 7; 

       unsigned char cur_val = dx[sbox_index[i]];

       cur_thread_partial_dy[sbox_index[i]] = TRIFLE::diff_table_shared[cur_val][diff_freq_index]; //Assigning target val to partial_dy

       prob_thread *= (TRIFLE::prob_table_shared[cur_val][diff_freq_index]);

       divide_factor *= 7;
   }
   
   if (sbox_num_local >  4){
       int s6,s7,s8;
       if (sbox_num_local == 5){
           s6 = 0;
           s7 = 0; 
           s8 = 0;
       }
       else if (sbox_num_local == 6){
           s6 = 6;
           s7 = 0; 
           s8 = 0;
       }
       else if (sbox_num_local==7){
           s6 = 6;
           s7 = 6; 
           s8 = 0;
       }
       else{
           s6 = 6;
           s7 = 6; 
           s8 = 6;
       }

       for (int s8_loop_i=0; s8_loop_i <= s8; s8_loop_i++){
           for (int i=4;i<sbox_num_local;i++){
               cur_thread_partial_dy[sbox_index[i]] = dx[sbox_index[i]];
           }    

           float cur_prob_final_fourth = prob_thread;
           unsigned char cur_thread_partial_dy_RN_fourth[33]; //The [32] or size 33 is for fast procesing only
           memcpy(cur_thread_partial_dy_RN_fourth, cur_thread_partial_dy, 32);
           cur_thread_partial_dy_RN_fourth[32] = 0;
   
           unsigned char cur_val = cur_thread_partial_dy_RN_fourth[sbox_index[7]];
           cur_thread_partial_dy_RN_fourth[sbox_index[7]] = TRIFLE::diff_table_shared[cur_val][s8_loop_i];
           cur_prob_final_fourth *= (TRIFLE::prob_table_shared[cur_val][s8_loop_i]);
       
           for (int s7_loop_i=0; s7_loop_i <= s7; s7_loop_i++){
               float cur_prob_final_third = cur_prob_final_fourth;
               unsigned char cur_thread_partial_dy_RN_third[33]; //The [32] or size 33 is for fast procesing only
               memcpy(cur_thread_partial_dy_RN_third, cur_thread_partial_dy_RN_fourth, 33);
               //cur_thread_partial_dy_RN_third[32] = 0; // No need as already copied over
   
               cur_val = cur_thread_partial_dy_RN_third[sbox_index[6]];
               cur_thread_partial_dy_RN_third[sbox_index[6]] = TRIFLE::diff_table_shared[cur_val][s7_loop_i];
               cur_prob_final_third *= (TRIFLE::prob_table_shared[cur_val][s7_loop_i]);

               for (int s6_loop_i=0; s6_loop_i <= s6; s6_loop_i++){
                   float cur_prob_final_second = cur_prob_final_third;
                   unsigned char cur_thread_partial_dy_RN_second[33]; //The [32] or size 33 is for fast procesing only
                   memcpy(cur_thread_partial_dy_RN_second, cur_thread_partial_dy_RN_third, 33);
   
                   cur_val = cur_thread_partial_dy_RN_second[sbox_index[5]];
                   cur_thread_partial_dy_RN_second[sbox_index[5]] = TRIFLE::diff_table_shared[cur_val][s6_loop_i];
                   cur_prob_final_second *= (TRIFLE::prob_table_shared[cur_val][s6_loop_i]);

                   for (int s5_loop_i=0;s5_loop_i<7;s5_loop_i++){
                       float cur_prob_final = cur_prob_final_second;
                       unsigned char cur_thread_partial_dy_RN[33]; //The [32] or size 33 is for fast procesing only
                       memcpy(cur_thread_partial_dy_RN, cur_thread_partial_dy_RN_second, 33);
   
                       //Substitution final
                       cur_val = cur_thread_partial_dy_RN[sbox_index[4]];
                       cur_thread_partial_dy_RN[sbox_index[4]] = TRIFLE::diff_table_shared[cur_val][s5_loop_i];
                       cur_prob_final *= (TRIFLE::prob_table_shared[cur_val][s5_loop_i] );    

                        //Permutation
                        unsigned char new_partial_dy[32] = { 0 };
                        unsigned long long front_64 = 0, back_64 = 0;

                        for (int i = 0; i < 32; i++) {
                            if (cur_thread_partial_dy_RN[i] > 0) {
                                front_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy_RN[i]][0];
                                back_64  |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy_RN[i]][1];
                            }
                        }
                        for (int i = 0; i < 16; i++) {
                            new_partial_dy[i] = (front_64 >> ((15 - i) * 4)) & 0xf;
                        }
                        for (int i = 16; i < 32; i++) {
                            new_partial_dy[i] = (back_64 >> ((31 - i) * 4)) & 0xf;
                        }

                        {
                            //Check for condition
                            int sbox_num=0;
                            int sbox_index[32]={0};
                            for (int i=0;i<32;i++){
                                if (new_partial_dy[i] !=0){
                                    sbox_index[sbox_num] = i;
                                    sbox_num+=1;
                                }
                            }

                            if (sbox_num <=3){ //Possible to store three only...
                                //Computing appropriate index
                                int index=0;
                                for (int i=0;i<sbox_num;i++){
                                    index|= ( ( (sbox_index[i]&0b11111) | ( (new_partial_dy[sbox_index[i]]&0b1111) << 5) ) << (i * 9) ); 
                                }

                                atomicAdd( MITM_size_interm_global+index, 1);
                                atomicAdd( MITM_prob_interm_global+index, cur_prob_final);
                            }
                        }
                   }
               }
           }
       }
   }
   else{
        //Permutation
        unsigned char new_partial_dy[32] = { 0 };
        unsigned long long front_64 = 0, back_64 = 0;

        for (int i = 0; i < 32; i++) {
            if (cur_thread_partial_dy[i] > 0) {
                front_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy[i]][0];
                back_64  |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy[i]][1];
            }
        }
        for (int i = 0; i < 16; i++) {
            new_partial_dy[i] = (front_64 >> ((15 - i) * 4)) & 0xf;
        }
        for (int i = 16; i < 32; i++) {
            new_partial_dy[i] = (back_64 >> ((31 - i) * 4)) & 0xf;
        }

        {
            //Check for condition
            int sbox_num=0;
            int sbox_index[32]={0};
            for (int i=0;i<32;i++){
                if (new_partial_dy[i] !=0){
                    sbox_index[sbox_num] = i;
                    sbox_num+=1;
                }
            }

            if (sbox_num <=3){ //Possible to store three only...
                //Computing appropriate index
                int index=0;
                for (int i=0;i<sbox_num;i++){
                    index|= ( ( (sbox_index[i]&0b11111) | ( (new_partial_dy[sbox_index[i]]&0b1111) << 5) ) << (i * 9) ); 
                }

                atomicAdd( MITM_size_interm_global+index, 1);
                atomicAdd( MITM_prob_interm_global+index, prob_thread);
            }
        }
   }
};

__global__ void kernel_trifle_n_backward(unsigned char* dx, int* sbox_num, int* sbox_index, float* cur_prob, float* dev_new_prob_arr,  unsigned int* dev_cluster_size_arr,
    float* MITM_prob_interm_global, int* MITM_size_interm_global, double* MITM_prob_final_global, long long* MITM_size_final_global){
    //Number of threads must be >=32
    if (threadIdx.x <32){
        if (threadIdx.x<16){
            TRIFLE::diff_table_size_shared[threadIdx.x] = TRIFLE::diff_table_size_global[threadIdx.x];

            for (int j = 0; j < 8; j++) {
                //NOTE: Reversed
                TRIFLE::diff_table_shared[threadIdx.x][j] = TRIFLE::diff_table_global_reversed[threadIdx.x][j];
                TRIFLE::prob_table_shared[threadIdx.x][j] = TRIFLE::freq_table_global[threadIdx.x][j]/16.0; 
            }
        }

        for (int j=0;j<16;j++){
            //NOTE: Reversed
            TRIFLE::perm_lookup_shared[threadIdx.x][j][0]= TRIFLE::perm_lookup_global_reversed[threadIdx.x][j][0];
            TRIFLE::perm_lookup_shared[threadIdx.x][j][1]= TRIFLE::perm_lookup_global_reversed[threadIdx.x][j][1];
        }
    }

   __syncthreads(); //wait for init to be finished, sync up all threads within a block... shared memory lies within each block.

   //Computing target array index (id and output_range)
   //I - THREAD ID / total thread (including all block) - Used to coordinate splitting of tasks
   int thread_id_global = (blockIdx.x * THREAD_PER_BLOCK) + threadIdx.x; 
   float prob_thread = (*cur_prob); //Caching into local memory
   int sbox_num_local = (*sbox_num); //NOTE: difference from n-1, where parameter is not pointer
   
   //Start Range
   // 2560 different threads  1-2559
   //Each thread is responsible for 2401 array entry in result array (n-1 rounds 2 rounds results)
   //There will be 256*10 = 2560 threads handling 2401 

   //0 - 2400
   int thread_id_workforce = 2401;
   if (thread_id_global >= thread_id_workforce) {
       return;
   } 

   //1 Round for 4AS and above
   unsigned char cur_thread_partial_dy[32] = { 0 };
   //Subs for 4 AS
   int divide_factor = 1;
   int diff_freq_index;
   int remaining_value = thread_id_global; //0 - 2400

   //NOTE: Only support sbox>=4
   for (int i = 0; i < 4; i++) {
       diff_freq_index = (remaining_value / divide_factor) % 7; 

       unsigned char cur_val = dx[sbox_index[i]];

       cur_thread_partial_dy[sbox_index[i]] = TRIFLE::diff_table_shared[cur_val][diff_freq_index]; //Assigning target val to partial_dy

       prob_thread *= (TRIFLE::prob_table_shared[cur_val][diff_freq_index]);

       divide_factor *= 7;
   }
   
   if (sbox_num_local >  4){
       int s6,s7,s8;
       if (sbox_num_local == 5){
           s6 = 0;
           s7 = 0; 
           s8 = 0;
       }
       else if (sbox_num_local == 6){
           s6 = 6;
           s7 = 0; 
           s8 = 0;
       }
       else if (sbox_num_local==7){
           s6 = 6;
           s7 = 6; 
           s8 = 0;
       }
       else{
           s6 = 6;
           s7 = 6; 
           s8 = 6;
       }

       for (int s8_loop_i=0; s8_loop_i <= s8; s8_loop_i++){
           for (int i=4;i<sbox_num_local;i++){
               cur_thread_partial_dy[sbox_index[i]] = dx[sbox_index[i]];
           }    

           float cur_prob_final_fourth = prob_thread;
           unsigned char cur_thread_partial_dy_RN_fourth[33]; //The [32] or size 33 is for fast procesing only
           memcpy(cur_thread_partial_dy_RN_fourth, cur_thread_partial_dy, 32);
           cur_thread_partial_dy_RN_fourth[32] = 0;
   
           unsigned char cur_val = cur_thread_partial_dy_RN_fourth[sbox_index[7]];
           cur_thread_partial_dy_RN_fourth[sbox_index[7]] = TRIFLE::diff_table_shared[cur_val][s8_loop_i];
           cur_prob_final_fourth *= (TRIFLE::prob_table_shared[cur_val][s8_loop_i]);
       
           for (int s7_loop_i=0; s7_loop_i <= s7; s7_loop_i++){
               float cur_prob_final_third = cur_prob_final_fourth;
               unsigned char cur_thread_partial_dy_RN_third[33]; //The [32] or size 33 is for fast procesing only
               memcpy(cur_thread_partial_dy_RN_third, cur_thread_partial_dy_RN_fourth, 33);
               //cur_thread_partial_dy_RN_third[32] = 0; // No need as already copied over
   
               cur_val = cur_thread_partial_dy_RN_third[sbox_index[6]];
               cur_thread_partial_dy_RN_third[sbox_index[6]] = TRIFLE::diff_table_shared[cur_val][s7_loop_i];
               cur_prob_final_third *= (TRIFLE::prob_table_shared[cur_val][s7_loop_i]);

               for (int s6_loop_i=0; s6_loop_i <= s6; s6_loop_i++){
                   float cur_prob_final_second = cur_prob_final_third;
                   unsigned char cur_thread_partial_dy_RN_second[33]; //The [32] or size 33 is for fast procesing only
                   memcpy(cur_thread_partial_dy_RN_second, cur_thread_partial_dy_RN_third, 33);
   
                   cur_val = cur_thread_partial_dy_RN_second[sbox_index[5]];
                   cur_thread_partial_dy_RN_second[sbox_index[5]] = TRIFLE::diff_table_shared[cur_val][s6_loop_i];
                   cur_prob_final_second *= (TRIFLE::prob_table_shared[cur_val][s6_loop_i]);

                   for (int s5_loop_i=0;s5_loop_i<7;s5_loop_i++){
                       float cur_prob_final = cur_prob_final_second;
                       unsigned char cur_thread_partial_dy_RN[33]; //The [32] or size 33 is for fast procesing only
                       memcpy(cur_thread_partial_dy_RN, cur_thread_partial_dy_RN_second, 33);
   
                       //Substitution final
                       cur_val = cur_thread_partial_dy_RN[sbox_index[4]];
                       cur_thread_partial_dy_RN[sbox_index[4]] = TRIFLE::diff_table_shared[cur_val][s5_loop_i];
                       cur_prob_final *= (TRIFLE::prob_table_shared[cur_val][s5_loop_i] );    

                        {
                            //Check for condition
                            int sbox_num=0;
                            int sbox_index[32]={0};
                            for (int i=0;i<32;i++){
                                if (cur_thread_partial_dy_RN[i] !=0){
                                    sbox_index[sbox_num] = i;
                                    sbox_num+=1;
                                }
                            }

                            if (sbox_num <=3){ //Possible to store three only...
                                //Computing appropriate index
                                int index=0;
                                for (int i=0;i<sbox_num;i++){
                                    index|= ( ( (sbox_index[i]&0b11111) | ( (cur_thread_partial_dy_RN[sbox_index[i]]&0b1111) << 5) ) << (i * 9) ); 
                                }

                                int target_size =  MITM_size_interm_global[index];
                                if(target_size > 0){ //Exist connection
                                    double target_prob = ( (double) cur_prob_final * MITM_prob_interm_global[index]);

                                    //Add to collection
                                    MITM_prob_final_global[thread_id_global] += target_prob;
                                    MITM_size_final_global[thread_id_global] += target_size;
                                }
                            }
                        }
                   }
               }
           }
       }
   }
   else{
        {
            //Check for condition
            int sbox_num=0;
            int sbox_index[32]={0};
            for (int i=0;i<32;i++){
                if (cur_thread_partial_dy[i] !=0){
                    sbox_index[sbox_num] = i;
                    sbox_num+=1;
                }
            }

            if (sbox_num <=3){ //Possible to store three only...
                //Computing appropriate index
                int index=0;
                for (int i=0;i<sbox_num;i++){
                    index|= ( ( (sbox_index[i]&0b11111) | ( (cur_thread_partial_dy[sbox_index[i]]&0b1111) << 5) ) << (i * 9) ); 
                }

                int target_size =  MITM_size_interm_global[index];
                if(target_size > 0){ //Exist connection
                    double target_prob = ( (double) prob_thread * MITM_prob_interm_global[index]);

                    //Add to collection
                    MITM_prob_final_global[thread_id_global] += target_prob;
                    MITM_size_final_global[thread_id_global] += target_size;
                }
            }
        }
   }
};

/*One round kernel compute for 0-n (not last round)
* Number of AS should be >=4 for optimal performance (AS with smaller than 4 is undefined behvaior)
* Input - DX, Sbox_num, Sbox_index, Cur_prob
* Output - Dev_dy_arr, dev_new_prob_arr
*/
__global__ void kernel_trifle_n_minus_one(unsigned char* dx, int* sbox_index, int sbox_num, float cur_prob, int cur_r, 
    unsigned char* dev_dy, float* dev_new_prob_arr, int* dev_sbox_num_and_index){
    //Number of threads must be >=32
    if (threadIdx.x <32){
        if (threadIdx.x<16){
            TRIFLE::diff_table_size_shared[threadIdx.x] = TRIFLE::diff_table_size_global[threadIdx.x];

            for (int j = 0; j < 8; j++) {
                TRIFLE::diff_table_shared[threadIdx.x][j] = TRIFLE::diff_table_global[threadIdx.x][j];
                TRIFLE::prob_table_shared[threadIdx.x][j] = TRIFLE::freq_table_global[threadIdx.x][j]/16.0; 
            }
        }

        for (int j=0;j<16;j++){
            TRIFLE::perm_lookup_shared[threadIdx.x][j][0]= TRIFLE::perm_lookup_global[threadIdx.x][j][0];
            TRIFLE::perm_lookup_shared[threadIdx.x][j][1]= TRIFLE::perm_lookup_global[threadIdx.x][j][1];
        }
    }

	__syncthreads(); //wait for init to be finished, sync up all threads within a block... shared memory lies within each block.

	//Computing target array index (id and output_range)
    //I - THREAD ID / total thread (including all block) - Used to coordinate splitting of tasks
    int thread_id_global = (blockIdx.x * THREAD_PER_BLOCK) + threadIdx.x; 
	float prob_thread = cur_prob; //Caching into local memory
	int sbox_num_local = sbox_num;

    int thread_process_amount = 1; //Amount of data to be processed by one thread
    int s5,s6,s7,s8;
    if (sbox_num_local == 4){
        thread_process_amount = 1;
    }
    else if (sbox_num_local == 5){
        thread_process_amount = 7;
        s5 = 6;
        s6 = 0;
        s7 = 0; 
        s8 = 0;
    }
    else if (sbox_num_local == 6){
        thread_process_amount = 49;
        s5 = 6;
        s6 = 6;
        s7 = 0; 
        s8 = 0;
    }
    else if (sbox_num_local==7){
        thread_process_amount = 343;
        s5 = 6;
        s6 = 6;
        s7 = 6; 
        s8 = 0;
    }
    else{
        thread_process_amount = 2401;
        s5 = 6;
        s6 = 6;
        s7 = 6; 
        s8 = 6;
    }

    unsigned char* output_dy = dev_dy + (thread_id_global*thread_process_amount*32);
    float* output_prob = dev_new_prob_arr + (thread_id_global*thread_process_amount*1);
    int* output_sbox_num = dev_sbox_num_and_index + (thread_id_global*thread_process_amount*9);
    int* output_sbox_index = output_sbox_num + 1; 

    //Start Range
	// 2560 different threads  1-2559
	//Each thread is responsible for 2401 array entry in result array (n-1 rounds 2 rounds results)
    //There will be 256*10 = 2560 threads handling 2401 

    //0 - 2400
    int thread_id_workforce = 2401;
	if (thread_id_global >= thread_id_workforce) {
		return;
    } 

	//1 Round for 4AS and above
	unsigned char cur_thread_partial_dy[32] = { 0 };
	//Subs for 4 AS
	{
		int divide_factor = 1;
		int diff_freq_index;
		int remaining_value = thread_id_global; //0 - 2400

        //NOTE: Only support sbox>=4
		for (int i = 0; i < 4; i++) {
			diff_freq_index = (remaining_value / divide_factor) % 7; 

			unsigned char cur_val = dx[sbox_index[i]];

			cur_thread_partial_dy[sbox_index[i]] = TRIFLE::diff_table_shared[cur_val][diff_freq_index]; //Assigning target val to partial_dy

			prob_thread *= (TRIFLE::prob_table_shared[cur_val][diff_freq_index]);

			divide_factor *= 7;
		}
    }

    if (sbox_num_local >  4){
        for (int i=4;i<sbox_num_local;i++){
            cur_thread_partial_dy[sbox_index[i]] = dx[sbox_index[i]];
        }

        for (int s8_loop_i=0; s8_loop_i <= s8; s8_loop_i++){
            float cur_prob_final_fourth = prob_thread;
            unsigned char cur_thread_partial_dy_RN_fourth[33]; //The [32] or size 33 is for fast procesing only
            memcpy(cur_thread_partial_dy_RN_fourth, cur_thread_partial_dy, 32);
            cur_thread_partial_dy_RN_fourth[32] = 0;
    
            unsigned char cur_val = cur_thread_partial_dy_RN_fourth[sbox_index[7]];
            cur_thread_partial_dy_RN_fourth[sbox_index[7]] = TRIFLE::diff_table_shared[cur_val][s8_loop_i];
            cur_prob_final_fourth *= (TRIFLE::prob_table_shared[cur_val][s8_loop_i]);
        
            for (int s7_loop_i=0; s7_loop_i <= s7; s7_loop_i++){
                float cur_prob_final_third = cur_prob_final_fourth;
                unsigned char cur_thread_partial_dy_RN_third[33]; //The [32] or size 33 is for fast procesing only
                memcpy(cur_thread_partial_dy_RN_third, cur_thread_partial_dy_RN_fourth, 33);
                //cur_thread_partial_dy_RN_third[32] = 0; // No need as already copied over
    
                cur_val = cur_thread_partial_dy_RN_third[sbox_index[6]];
                cur_thread_partial_dy_RN_third[sbox_index[6]] = TRIFLE::diff_table_shared[cur_val][s7_loop_i];
                cur_prob_final_third *= (TRIFLE::prob_table_shared[cur_val][s7_loop_i]);

                for (int s6_loop_i=0; s6_loop_i <= s6; s6_loop_i++){
                    float cur_prob_final_second = cur_prob_final_third;
                    unsigned char cur_thread_partial_dy_RN_second[33]; //The [32] or size 33 is for fast procesing only
                    memcpy(cur_thread_partial_dy_RN_second, cur_thread_partial_dy_RN_third, 33);
    
                    cur_val = cur_thread_partial_dy_RN_second[sbox_index[5]];
                    cur_thread_partial_dy_RN_second[sbox_index[5]] = TRIFLE::diff_table_shared[cur_val][s6_loop_i];
                    cur_prob_final_second *= (TRIFLE::prob_table_shared[cur_val][s6_loop_i]);

                    for (int s5_loop_i=0;s5_loop_i<7;s5_loop_i++){
                        float cur_prob_final = cur_prob_final_second;
                        unsigned char cur_thread_partial_dy_RN[33]; //The [32] or size 33 is for fast procesing only
                        memcpy(cur_thread_partial_dy_RN, cur_thread_partial_dy_RN_second, 33);
    
                        //Substitution final
                        cur_val = cur_thread_partial_dy_RN[sbox_index[4]];
                        cur_thread_partial_dy_RN[sbox_index[4]] = TRIFLE::diff_table_shared[cur_val][s5_loop_i];
                        cur_prob_final *= (TRIFLE::prob_table_shared[cur_val][s5_loop_i] );    

                        //Permutation
                        unsigned long long front_64 = 0, back_64 = 0;
                        for (int i = 0; i < 32; i++) {
                            if ( cur_thread_partial_dy_RN[i] > 0) {
                                //Permutation LUTable
                                //25% less running time compared to normal computation
                                front_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy_RN[i]][0];
                                back_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy_RN[i]][1];
                            }
                        }
                        #pragma unroll
                        for (int i = 0; i < 16; i++) {
                            cur_thread_partial_dy_RN[i] = (front_64 >> ((15 - i) * 4)) & 0xf;
                        }
                        #pragma unroll
                        for (int i = 16; i < 32; i++) {
                            cur_thread_partial_dy_RN[i] = (back_64 >> ((31 - i) * 4)) & 0xf;
                        }
                        //cur_thread_partial_dy is already subbed and permutated..
                        //prob_thread records its probabilities

                        //Calculte sbox index and sbox number
                        int cur_sbox_num = 0;
                        int cur_sbox_index[32]; //Will point to non existance 32 array entry (see substitution below)
                        #pragma unroll
                        for (int i=0;i<8;i++){
                            cur_sbox_index[i] = 32;
                        }
                        #pragma unroll
                        for (int i = 0; i < 32; i++) {
                            if ((cur_thread_partial_dy_RN[i] & 0xf) > 0) {
                                cur_sbox_index[cur_sbox_num] = i;
                                cur_sbox_num++;
                            }
                        }

                        //Calculate Matsui Bound
                        if (cur_sbox_num <= MAX_AS_USED)  //If only next round AS <= 8
                        {
                            //MATSUI BOUND
                            float estimated_com_prob = ( powf(CLUSTER_PROB_INDIV, (PATTERN_ROUND_MITM_FORWARD - cur_r - 2)) * powf(CLUSTER_1AS_BEST_PROB, cur_sbox_num) );
                            if ((estimated_com_prob * cur_prob_final) >= TRIFLE::CLUSTER_PROB_BOUND_const) {
                            // if ((estimated_com_prob * cur_prob_final) >= TRIFLE::CLUSTER_PROB_BOUND_const) {
                            // if (true) {
                                //Save everything
                                memcpy(output_dy,cur_thread_partial_dy_RN,32);
                                *output_prob = cur_prob_final;
                                *output_sbox_num = cur_sbox_num;
                                memcpy(output_sbox_index, cur_sbox_index, sizeof(int) * 8 );
                            } else{
                                *output_sbox_num = 0; //Indicate jump over this
                            }
                        }
                        else{
                            *output_sbox_num = 0; //Indicate jump over this
                        }

                        //Calculate the next set of address to save to
                        output_dy = output_dy + 32;
                        output_prob = output_prob + 1;
                        output_sbox_num = output_sbox_num + 9;
                        output_sbox_index = output_sbox_index + 9;
                    }
                }
            }
        }
    }
    else{
        //Permutation
        unsigned long long front_64 = 0, back_64 = 0;
        for (int i = 0; i < 32; i++) {
            if ( cur_thread_partial_dy[i] > 0) {
                //Permutation LUTable
                //25% less running time compared to normal computation
                front_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy[i]][0];
                back_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy[i]][1];
            }
        }
        #pragma unroll
        for (int i = 0; i < 16; i++) {
            cur_thread_partial_dy[i] = (front_64 >> ((15 - i) * 4)) & 0xf;
        }
        #pragma unroll
        for (int i = 16; i < 32; i++) {
            cur_thread_partial_dy[i] = (back_64 >> ((31 - i) * 4)) & 0xf;
        }
        //cur_thread_partial_dy is already subbed and permutated..
        //prob_thread records its probabilities
        
        //Calculte sbox index and sbox number
        int cur_sbox_num = 0;
        int cur_sbox_index[32]; //Will point to non existance 32 array entry (see substitution below)
        for (int i=0;i<8;i++){
            cur_sbox_index[i] = 32;
        }
        for (int i = 0; i < 32; i++) {
            if ((cur_thread_partial_dy[i] & 0xf) > 0) {
                cur_sbox_index[cur_sbox_num] = i;
                cur_sbox_num++;
            }
        }

        //Calculate Matsui Bound
        if (cur_sbox_num <= MAX_AS_USED)  //If only next round AS <= 8
        {
            //MATSUI BOUND
            float estimated_com_prob = ( powf(CLUSTER_PROB_INDIV, (PATTERN_ROUND_MITM_FORWARD - cur_r - 2)) * powf(CLUSTER_1AS_BEST_PROB, cur_sbox_num) );
            if ((estimated_com_prob * prob_thread) >= TRIFLE::CLUSTER_PROB_BOUND_const) {
            // if (true) {
                //Save everything
                memcpy(output_dy,cur_thread_partial_dy,32);
                *output_prob = prob_thread;
                *output_sbox_num = cur_sbox_num;
                memcpy(output_sbox_index, cur_sbox_index, sizeof(int) * 8 );
            } else{
                *output_sbox_num = 0; //Indicate jump over this
            }
        }
        else{
            *output_sbox_num = 0; //Indicate jump over this
        }
    }
};

__global__ void kernel_trifle_n_minus_one_backward(unsigned char* dx, int* sbox_index, int sbox_num, float cur_prob, int cur_r, 
    unsigned char* dev_dy, float* dev_new_prob_arr, int* dev_sbox_num_and_index){
    //NOTE: Different between this and normal one is only during copying to shared variable

    //Number of threads must be >=32
    if (threadIdx.x <32){
        if (threadIdx.x<16){
            TRIFLE::diff_table_size_shared[threadIdx.x] = TRIFLE::diff_table_size_global[threadIdx.x];

            for (int j = 0; j < 8; j++) {
                //NOTE: Reversed
                TRIFLE::diff_table_shared[threadIdx.x][j] = TRIFLE::diff_table_global_reversed[threadIdx.x][j];
                TRIFLE::prob_table_shared[threadIdx.x][j] = TRIFLE::freq_table_global[threadIdx.x][j]/16.0; 
            }
        }

        for (int j=0;j<16;j++){
            //NOTE: Reversed
            TRIFLE::perm_lookup_shared[threadIdx.x][j][0]= TRIFLE::perm_lookup_global_reversed[threadIdx.x][j][0];
            TRIFLE::perm_lookup_shared[threadIdx.x][j][1]= TRIFLE::perm_lookup_global_reversed[threadIdx.x][j][1];
        }
    }

	__syncthreads(); //wait for init to be finished, sync up all threads within a block... shared memory lies within each block.

	//Computing target array index (id and output_range)
    //I - THREAD ID / total thread (including all block) - Used to coordinate splitting of tasks
    int thread_id_global = (blockIdx.x * THREAD_PER_BLOCK) + threadIdx.x; 
	float prob_thread = cur_prob; //Caching into local memory
	int sbox_num_local = sbox_num;

    int thread_process_amount = 1; //Amount of data to be processed by one thread
    int s5,s6,s7,s8;
    if (sbox_num_local == 4){
        thread_process_amount = 1;
    }
    else if (sbox_num_local == 5){
        thread_process_amount = 7;
        s5 = 6;
        s6 = 0;
        s7 = 0; 
        s8 = 0;
    }
    else if (sbox_num_local == 6){
        thread_process_amount = 49;
        s5 = 6;
        s6 = 6;
        s7 = 0; 
        s8 = 0;
    }
    else if (sbox_num_local==7){
        thread_process_amount = 343;
        s5 = 6;
        s6 = 6;
        s7 = 6; 
        s8 = 0;
    }
    else{
        thread_process_amount = 2401;
        s5 = 6;
        s6 = 6;
        s7 = 6; 
        s8 = 6;
    }

    unsigned char* output_dy = dev_dy + (thread_id_global*thread_process_amount*32);
    float* output_prob = dev_new_prob_arr + (thread_id_global*thread_process_amount*1);
    int* output_sbox_num = dev_sbox_num_and_index + (thread_id_global*thread_process_amount*9);
    int* output_sbox_index = output_sbox_num + 1; 

    //Start Range
	// 2560 different threads  1-2559
	//Each thread is responsible for 2401 array entry in result array (n-1 rounds 2 rounds results)
    //There will be 256*10 = 2560 threads handling 2401 

    //0 - 2400
    int thread_id_workforce = 2401;
	if (thread_id_global >= thread_id_workforce) {
		return;
    } 

	//1 Round for 4AS and above
	unsigned char cur_thread_partial_dy[32] = { 0 };
	//Subs for 4 AS
	{
		int divide_factor = 1;
		int diff_freq_index;
		int remaining_value = thread_id_global; //0 - 2400

        //NOTE: Only support sbox>=4
		for (int i = 0; i < 4; i++) {
			diff_freq_index = (remaining_value / divide_factor) % 7; 

			unsigned char cur_val = dx[sbox_index[i]];

			cur_thread_partial_dy[sbox_index[i]] = TRIFLE::diff_table_shared[cur_val][diff_freq_index]; //Assigning target val to partial_dy

			prob_thread *= (TRIFLE::prob_table_shared[cur_val][diff_freq_index]);

			divide_factor *= 7;
		}
    }

    if (sbox_num_local >  4){
        for (int i=4;i<sbox_num_local;i++){
            cur_thread_partial_dy[sbox_index[i]] = dx[sbox_index[i]];
        }

        for (int s8_loop_i=0; s8_loop_i <= s8; s8_loop_i++){
            float cur_prob_final_fourth = prob_thread;
            unsigned char cur_thread_partial_dy_RN_fourth[33]; //The [32] or size 33 is for fast procesing only
            memcpy(cur_thread_partial_dy_RN_fourth, cur_thread_partial_dy, 32);
            cur_thread_partial_dy_RN_fourth[32] = 0;
    
            unsigned char cur_val = cur_thread_partial_dy_RN_fourth[sbox_index[7]];
            cur_thread_partial_dy_RN_fourth[sbox_index[7]] = TRIFLE::diff_table_shared[cur_val][s8_loop_i];
            cur_prob_final_fourth *= (TRIFLE::prob_table_shared[cur_val][s8_loop_i]);
        
            for (int s7_loop_i=0; s7_loop_i <= s7; s7_loop_i++){
                float cur_prob_final_third = cur_prob_final_fourth;
                unsigned char cur_thread_partial_dy_RN_third[33]; //The [32] or size 33 is for fast procesing only
                memcpy(cur_thread_partial_dy_RN_third, cur_thread_partial_dy_RN_fourth, 33);
                //cur_thread_partial_dy_RN_third[32] = 0; // No need as already copied over
    
                cur_val = cur_thread_partial_dy_RN_third[sbox_index[6]];
                cur_thread_partial_dy_RN_third[sbox_index[6]] = TRIFLE::diff_table_shared[cur_val][s7_loop_i];
                cur_prob_final_third *= (TRIFLE::prob_table_shared[cur_val][s7_loop_i]);

                for (int s6_loop_i=0; s6_loop_i <= s6; s6_loop_i++){
                    float cur_prob_final_second = cur_prob_final_third;
                    unsigned char cur_thread_partial_dy_RN_second[33]; //The [32] or size 33 is for fast procesing only
                    memcpy(cur_thread_partial_dy_RN_second, cur_thread_partial_dy_RN_third, 33);
    
                    cur_val = cur_thread_partial_dy_RN_second[sbox_index[5]];
                    cur_thread_partial_dy_RN_second[sbox_index[5]] = TRIFLE::diff_table_shared[cur_val][s6_loop_i];
                    cur_prob_final_second *= (TRIFLE::prob_table_shared[cur_val][s6_loop_i]);

                    for (int s5_loop_i=0;s5_loop_i<7;s5_loop_i++){
                        float cur_prob_final = cur_prob_final_second;
                        unsigned char cur_thread_partial_dy_RN[33]; //The [32] or size 33 is for fast procesing only
                        memcpy(cur_thread_partial_dy_RN, cur_thread_partial_dy_RN_second, 33);
    
                        //Substitution final
                        cur_val = cur_thread_partial_dy_RN[sbox_index[4]];
                        cur_thread_partial_dy_RN[sbox_index[4]] = TRIFLE::diff_table_shared[cur_val][s5_loop_i];
                        cur_prob_final *= (TRIFLE::prob_table_shared[cur_val][s5_loop_i] );    

                        //Permutation
                        unsigned long long front_64 = 0, back_64 = 0;
                        for (int i = 0; i < 32; i++) {
                            if ( cur_thread_partial_dy_RN[i] > 0) {
                                //Permutation LUTable
                                //25% less running time compared to normal computation
                                front_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy_RN[i]][0];
                                back_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy_RN[i]][1];
                            }
                        }
                        #pragma unroll
                        for (int i = 0; i < 16; i++) {
                            cur_thread_partial_dy_RN[i] = (front_64 >> ((15 - i) * 4)) & 0xf;
                        }
                        #pragma unroll
                        for (int i = 16; i < 32; i++) {
                            cur_thread_partial_dy_RN[i] = (back_64 >> ((31 - i) * 4)) & 0xf;
                        }
                        //cur_thread_partial_dy is already subbed and permutated..
                        //prob_thread records its probabilities

                        //Calculte sbox index and sbox number
                        int cur_sbox_num = 0;
                        int cur_sbox_index[32]; //Will point to non existance 32 array entry (see substitution below)
                        #pragma unroll
                        for (int i=0;i<8;i++){
                            cur_sbox_index[i] = 32;
                        }
                        #pragma unroll
                        for (int i = 0; i < 32; i++) {
                            if ((cur_thread_partial_dy_RN[i] & 0xf) > 0) {
                                cur_sbox_index[cur_sbox_num] = i;
                                cur_sbox_num++;
                            }
                        }

                        //Calculate Matsui Bound
                        if (cur_sbox_num <= MAX_AS_USED)  //If only next round AS <= 8
                        {
                            //MATSUI BOUND
                            float estimated_com_prob = ( powf(CLUSTER_PROB_INDIV, (PATTERN_ROUND_MITM_BACKWARD - cur_r - 2)) * powf(CLUSTER_1AS_BEST_PROB, cur_sbox_num) );
                            if ((estimated_com_prob * cur_prob_final) >= TRIFLE::CLUSTER_PROB_BOUND_const) {
                            // if ((estimated_com_prob * cur_prob_final) >= TRIFLE::CLUSTER_PROB_BOUND_const) {
                            // if (true) {
                                //Save everything
                                memcpy(output_dy,cur_thread_partial_dy_RN,32);
                                *output_prob = cur_prob_final;
                                *output_sbox_num = cur_sbox_num;
                                memcpy(output_sbox_index, cur_sbox_index, sizeof(int) * 8 );
                            } else{
                                *output_sbox_num = 0; //Indicate jump over this
                            }
                        }
                        else{
                            *output_sbox_num = 0; //Indicate jump over this
                        }

                        //Calculate the next set of address to save to
                        output_dy = output_dy + 32;
                        output_prob = output_prob + 1;
                        output_sbox_num = output_sbox_num + 9;
                        output_sbox_index = output_sbox_index + 9;
                    }
                }
            }
        }
    }
    else{
        //Permutation
        unsigned long long front_64 = 0, back_64 = 0;
        for (int i = 0; i < 32; i++) {
            if ( cur_thread_partial_dy[i] > 0) {
                //Permutation LUTable
                //25% less running time compared to normal computation
                front_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy[i]][0];
                back_64 |= TRIFLE::perm_lookup_shared[i][cur_thread_partial_dy[i]][1];
            }
        }
        #pragma unroll
        for (int i = 0; i < 16; i++) {
            cur_thread_partial_dy[i] = (front_64 >> ((15 - i) * 4)) & 0xf;
        }
        #pragma unroll
        for (int i = 16; i < 32; i++) {
            cur_thread_partial_dy[i] = (back_64 >> ((31 - i) * 4)) & 0xf;
        }
        //cur_thread_partial_dy is already subbed and permutated..
        //prob_thread records its probabilities
        
        //Calculte sbox index and sbox number
        int cur_sbox_num = 0;
        int cur_sbox_index[32]; //Will point to non existance 32 array entry (see substitution below)
        for (int i=0;i<8;i++){
            cur_sbox_index[i] = 32;
        }
        for (int i = 0; i < 32; i++) {
            if ((cur_thread_partial_dy[i] & 0xf) > 0) {
                cur_sbox_index[cur_sbox_num] = i;
                cur_sbox_num++;
            }
        }

        //Calculate Matsui Bound
        if (cur_sbox_num <= MAX_AS_USED)  //If only next round AS <= 8
        {
            //MATSUI BOUND
            float estimated_com_prob = ( powf(CLUSTER_PROB_INDIV, (PATTERN_ROUND_MITM_BACKWARD - cur_r - 2)) * powf(CLUSTER_1AS_BEST_PROB, cur_sbox_num) );
            if ((estimated_com_prob * prob_thread) >= TRIFLE::CLUSTER_PROB_BOUND_const) {
            // if (true) {
                //Save everything
                memcpy(output_dy,cur_thread_partial_dy,32);
                *output_prob = prob_thread;
                *output_sbox_num = cur_sbox_num;
                memcpy(output_sbox_index, cur_sbox_index, sizeof(int) * 8 );
            } else{
                *output_sbox_num = 0; //Indicate jump over this
            }
        }
        else{
            *output_sbox_num = 0; //Indicate jump over this
        }
    }
};

void Kernel_TRIFLE_t::kernel_compute_1round(unsigned char* dx, int* sbox_index, int sbox_num, float cur_prob, int cur_r,
    unsigned char* next_round_dx_array_pinned, float* cur_round_prob_pinned, int* next_round_sbox_num_and_index){
    hipError_t cudaStatus;

    //Input Copy
    cudaStatus = hipMemcpyAsync(device_dx, dx, sizeof(unsigned char) * 32, hipMemcpyHostToDevice, (this->stream_obj) );
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync (next_round_dx_array_pinned) failed!");
        }
    #endif

    cudaStatus = hipMemcpyAsync(device_sbox_index, sbox_index, sizeof(int) * 8, hipMemcpyHostToDevice, (this->stream_obj) );
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync (sbox_index) failed!");
        }
    #endif

    //Grid Size (Num Block), Block Size, Shared Memory dynamically allocated, stream 
    kernel_trifle_n_minus_one<<<BLOCK_NUM, THREAD_PER_BLOCK, 0, (this->stream_obj)>>> (this->device_dx, this->device_sbox_index, sbox_num, cur_prob, cur_r,
        this->device_dy_arr, this->device_prob_arr, this->device_sbox_num_and_index_arr);

    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error in kernel launch: %s - %s\n",  hipGetErrorName(cudaStatus), hipGetErrorString(cudaStatus));
      exit(-1);
    }
    
    //Output Copy
    //How much to copy out depends on sbox_num

    // int total_branch = std::pow(7, sbox_num);

    // cudaStatus = hipMemcpyAsync(next_round_dx_array_pinned, device_dy_arr, sizeof(unsigned char) * 32 * total_branch, hipMemcpyDeviceToHost, (this->stream_obj) );
    // #ifdef CUDA_ERROR_PRINT
    //     if (cudaStatus != hipSuccess) {
    //         fprintf(stderr, "hipMemcpyAsync (next_round_dx_array_pinned) failed!");
    //     }
    // #endif

    // cudaStatus = hipMemcpyAsync(cur_round_prob_pinned, device_prob_arr, sizeof(float) * total_branch, hipMemcpyDeviceToHost, (this->stream_obj) );
    // #ifdef CUDA_ERROR_PRINT
    //     if (cudaStatus != hipSuccess) {
    //         fprintf(stderr, "hipMemcpyAsync (cur_round_prob_pinned) failed!");
    //     }
    // #endif

    // cudaStatus = hipMemcpyAsync(next_round_sbox_num_and_index, device_sbox_num_and_index_arr, sizeof(int) * 9 * total_branch, hipMemcpyDeviceToHost, (this->stream_obj) );
    // #ifdef CUDA_ERROR_PRINT
    //     if (cudaStatus != hipSuccess) {
    //         fprintf(stderr, "hipMemcpyAsync (next_round_sbox_num_and_index) failed!");
    //     }
    // #endif


    //Wait until complete
    cudaStatus = hipStreamSynchronize(this->stream_obj);
    if(cudaStatus != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(cudaStatus));
      exit(-1);
    }
};

void Kernel_TRIFLE_t::kernel_compute_1round_backward(unsigned char* dx, int* sbox_index, int sbox_num, float cur_prob, int cur_r,
    unsigned char* next_round_dx_array_pinned, float* cur_round_prob_pinned, int* next_round_sbox_num_and_index){
    hipError_t cudaStatus;

    //Input Copy
    cudaStatus = hipMemcpyAsync(device_dx, dx, sizeof(unsigned char) * 32, hipMemcpyHostToDevice, (this->stream_obj) );
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync (next_round_dx_array_pinned) failed!");
        }
    #endif

    cudaStatus = hipMemcpyAsync(device_sbox_index, sbox_index, sizeof(int) * 8, hipMemcpyHostToDevice, (this->stream_obj) );
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync (sbox_index) failed!");
        }
    #endif

    //Grid Size (Num Block), Block Size, Shared Memory dynamically allocated, stream 
    kernel_trifle_n_minus_one_backward<<<BLOCK_NUM, THREAD_PER_BLOCK, 0, (this->stream_obj)>>> (this->device_dx, this->device_sbox_index, sbox_num, cur_prob, cur_r,
        this->device_dy_arr, this->device_prob_arr, this->device_sbox_num_and_index_arr);

    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error in kernel launch: %s - %s\n",  hipGetErrorName(cudaStatus), hipGetErrorString(cudaStatus));
      exit(-1);
    }
    
    //Output Copy
    //How much to copy out depends on sbox_num
    int total_branch = std::pow(7, sbox_num);

    cudaStatus = hipMemcpyAsync(next_round_dx_array_pinned, device_dy_arr, sizeof(unsigned char) * 32 * total_branch, hipMemcpyDeviceToHost, (this->stream_obj) );
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync (next_round_dx_array_pinned) failed!");
        }
    #endif

    cudaStatus = hipMemcpyAsync(cur_round_prob_pinned, device_prob_arr, sizeof(float) * total_branch, hipMemcpyDeviceToHost, (this->stream_obj) );
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync (cur_round_prob_pinned) failed!");
        }
    #endif

    cudaStatus = hipMemcpyAsync(next_round_sbox_num_and_index, device_sbox_num_and_index_arr, sizeof(int) * 9 * total_branch, hipMemcpyDeviceToHost, (this->stream_obj) );
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync (next_round_sbox_num_and_index) failed!");
        }
    #endif

    //Wait until complete
    cudaStatus = hipStreamSynchronize(this->stream_obj);
    if(cudaStatus != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(cudaStatus));
      exit(-1);
    }
};

void Kernel_TRIFLE_t::kernel_compute_1round_last(int offset_index){
    //unsigned char* dx, int* sbox_num, int* sbox_index, float* cur_prob, float* dev_new_prob_arr, int* dev_cluster_size_arr
    // ( device_dy_arr+(offset_index * 32), device_sbox_num_and_index_arr + (offset_index * 9) 
    // , device_sbox_num_and_index_arr+1+(offset_index*9), device_prob_arr+(offset_index)
    // , this->device_prob_final ,this->device_cluster_size_final );

    kernel_trifle_n<<<BLOCK_NUM, THREAD_PER_BLOCK, 0, (this->stream_obj)>>> 
        ( device_dy_arr+(offset_index * 32), device_sbox_num_and_index_arr + (offset_index * 9) 
            , device_sbox_num_and_index_arr+(offset_index*9)+1, device_prob_arr+(offset_index)
            , this->device_prob_final ,this->device_cluster_size_final );

    auto cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error in kernel launch: %s - %s\n",  hipGetErrorName(cudaStatus), hipGetErrorString(cudaStatus));
        exit(-1);
    }
};

void Kernel_TRIFLE_t::MITM_kernel_compute_1round_last_forward(int offset_index){

    kernel_trifle_n_forward<<<BLOCK_NUM, THREAD_PER_BLOCK, 0, (this->stream_obj)>>> 
        ( device_dy_arr+(offset_index * 32), device_sbox_num_and_index_arr + (offset_index * 9) 
            , device_sbox_num_and_index_arr+(offset_index*9)+1, device_prob_arr+(offset_index)
            , this->device_prob_final ,this->device_cluster_size_final ,MITM_prob_interm_global,MITM_size_interm_global 
            );
        
    auto cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error in kernel launch: %s - %s\n",  hipGetErrorName(cudaStatus), hipGetErrorString(cudaStatus));
        exit(-1);
    }
}

void Kernel_TRIFLE_t::MITM_kernel_compute_1round_last_backward(int offset_index)
{
    kernel_trifle_n_backward<<<BLOCK_NUM, THREAD_PER_BLOCK, 0, (this->stream_obj)>>> 
        ( device_dy_arr+(offset_index * 32), device_sbox_num_and_index_arr + (offset_index * 9) 
            , device_sbox_num_and_index_arr+(offset_index*9)+1, device_prob_arr+(offset_index)
            , this->device_prob_final ,this->device_cluster_size_final
            , MITM_prob_interm_global,MITM_size_interm_global 
            , MITM_prob_final_global, MITM_size_final_global );

    auto cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error in kernel launch: %s - %s\n",  hipGetErrorName(cudaStatus), hipGetErrorString(cudaStatus));
        exit(-1);
    }
}

void Kernel_TRIFLE_t::kernel_MITM_intermediate_reduction(float* prob_interm, int* size_interm){
    float* prob_staging = new float[MITM_size];
    int* size_staging = new int[MITM_size];

    auto cudaStatus = hipMemcpy(prob_staging, MITM_prob_interm_global, sizeof(float)* MITM_size, hipMemcpyDeviceToHost);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (next_round_dx_array_pinned) failed!");\
            getchar();
        }
    #endif

    cudaStatus = hipMemcpy(size_staging, MITM_size_interm_global, sizeof(int) * MITM_size, hipMemcpyDeviceToHost);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (cur_round_prob_pinned) failed!");
            getchar();
        }
    #endif

    for (int i=0;i< MITM_size;i++){ //Automatically modify back to the source
        prob_interm[i] += prob_staging[i];
        size_interm[i] += size_staging[i];
    }

    cudaStatus = hipMemcpy(MITM_prob_interm_global, prob_interm, sizeof(float) * MITM_size, hipMemcpyHostToDevice);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (cur_round_prob_pinned) failed!");
            getchar();
        }
    #endif

    cudaStatus = hipMemcpy(MITM_size_interm_global, size_interm, sizeof(int) * MITM_size, hipMemcpyHostToDevice);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (cur_round_prob_pinned) failed!");
            getchar();
        }
    #endif

    delete[] prob_staging;
    delete[] size_staging; 
}


void Kernel_TRIFLE_t::kernel_reduction(long long& gpu_cluster_size, double &gpu_cluster_prob){
    unsigned int device_cluster_size_final_arr[THREAD_PER_BLOCK * BLOCK_NUM]; //unsigned int * thread num * thread block 
    float device_prob_final_arr[THREAD_PER_BLOCK * BLOCK_NUM];       //float * thread_num * thread_block

    long long MITM_final_size_arr[THREAD_PER_BLOCK * BLOCK_NUM];
    double MITM_final_prob_arr[THREAD_PER_BLOCK * BLOCK_NUM];

    auto cudaStatus = hipMemcpy(device_cluster_size_final_arr, device_cluster_size_final, sizeof(unsigned int)* THREAD_PER_BLOCK*BLOCK_NUM, hipMemcpyDeviceToHost);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (next_round_dx_array_pinned) failed!");\
            getchar();
        }
    #endif

    cudaStatus = hipMemcpy(device_prob_final_arr, device_prob_final, sizeof(float) * THREAD_PER_BLOCK*BLOCK_NUM, hipMemcpyDeviceToHost);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (cur_round_prob_pinned) failed!");
            getchar();
        }
    #endif

    for (int i=0;i<(THREAD_PER_BLOCK*BLOCK_NUM); i++ ){
        gpu_cluster_size += device_cluster_size_final_arr[i];
        gpu_cluster_prob += device_prob_final_arr[i];
    }

    cudaStatus = hipMemcpy(MITM_final_size_arr, MITM_size_final_global, sizeof(long long)* THREAD_PER_BLOCK*BLOCK_NUM, hipMemcpyDeviceToHost);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (next_round_dx_array_pinned) failed!");\
            getchar();
        }
    #endif

    cudaStatus = hipMemcpy(MITM_final_prob_arr, MITM_prob_final_global, sizeof(double) * THREAD_PER_BLOCK*BLOCK_NUM, hipMemcpyDeviceToHost);
    #ifdef CUDA_ERROR_PRINT
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (cur_round_prob_pinned) failed!");
            getchar();
        }
    #endif

    for (int i=0;i<(THREAD_PER_BLOCK*BLOCK_NUM); i++ ){
        gpu_cluster_size += MITM_final_size_arr[i];
        gpu_cluster_prob += MITM_final_prob_arr[i];
    }

}   

void Kernel_TRIFLE_t::change_parameter(unsigned char* new_dy, unsigned char* new_dx){
    //Set DX and DY
    auto cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(TRIFLE::final_dy_constant), new_dy, sizeof(unsigned char)*32);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol final_dy_constant failed!");
		goto Error;
    }
    
    memcpy(TRIFLE::final_dy_host,new_dy,32);
    if ( new_dx!=nullptr ){
        memcpy(TRIFLE::ref_dx_host,new_dx,32);
    }

    //Reset Result
    cudaStatus = hipMemset(device_prob_final, 0, sizeof(float)*  THREAD_PER_BLOCK * BLOCK_NUM);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset device_prob_final failed!");
		goto Error;
    }

    cudaStatus = hipMemset(device_cluster_size_final, 0, sizeof(unsigned int)* THREAD_PER_BLOCK * BLOCK_NUM);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset device_cluster_size_final failed!");
		goto Error;
    }

    cudaStatus = hipMemset(MITM_prob_interm_global, 0, sizeof(float)*  Kernel_TRIFLE_t::MITM_size );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_prob_interm_global failed!");
		goto Error;
    }

    cudaStatus = hipMemset(MITM_size_interm_global, 0, sizeof(int)*  Kernel_TRIFLE_t::MITM_size );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_size_interm_global failed!");
		goto Error;
    }

    cudaStatus = hipMemset(MITM_prob_final_global, 0, sizeof(double)*  THREAD_PER_BLOCK*BLOCK_NUM );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_prob_final_global failed!");
		goto Error;
    }

    cudaStatus = hipMemset(MITM_size_final_global, 0, sizeof(long long)*  THREAD_PER_BLOCK*BLOCK_NUM );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset MITM_size_final_global failed!");
		goto Error;
    }
    
    return;

    Error:
    return;
}
